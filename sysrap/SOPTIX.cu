#include "hip/hip_runtime.h"
/**
SOPTIX.cu
===========


Functions
-----------

trace
    populate quad2 prd by call to optixTrace

make_normal_pixel
    minimal normal "shader"

render
    raygen function : calling trace and "shading" pixels

__raygen__rg
    calls one of the above raygen functions depending on params.raygenmode

setPayload
    mechanics of communication when not using WITH_PRD

__miss_ms
    default quad2 prd OR payload for rays that miss

__closesthit__ch
    populate quad2 prd OR payload for rays that intersect

**/

#include <optix.h>
#include <stdint.h>

#include "scuda.h"
#include "squad.h"

#include "SOPTIX_Binding.h"
#include "SOPTIX_Params.h"

#include "scuda_pointer.h"
#include "SOPTIX_getPRD.h"

extern "C" { __constant__ SOPTIX_Params params ;  }

/**
trace : pure function, with no use of params, everything via args
-------------------------------------------------------------------

Outcome of trace is to populate *prd* by payload and attribute passing.
When WITH_PRD macro is defined only 2 32-bit payload values are used to
pass the 64-bit  pointer, otherwise more payload and attributes values
are used to pass the contents IS->CH->RG.

See __closesthit__ch to see where the payload p0-p7 comes from.
**/

static __forceinline__ __device__ void trace(
        OptixTraversableHandle handle,
        float3                 ray_origin,
        float3                 ray_direction,
        float                  tmin,
        float                  tmax,
        quad2*                 prd,
        unsigned               visibilityMask
        )
{
    const float rayTime = 0.0f ;
    OptixRayFlags rayFlags = OPTIX_RAY_FLAG_DISABLE_ANYHIT ;   // OPTIX_RAY_FLAG_NONE
    const unsigned SBToffset = 0u ;
    const unsigned SBTstride = 1u ;
    const unsigned missSBTIndex = 0u ;
    uint32_t p0, p1 ;
    packPointer( prd, p0, p1 );  // scuda_pointer.h : pack prd addr from RG program into two uint32_t passed as payload
    optixTrace(
            handle,
            ray_origin,
            ray_direction,
            tmin,
            tmax,
            rayTime,
            visibilityMask,
            rayFlags,
            SBToffset,
            SBTstride,
            missSBTIndex,
            p0, p1
            );
}


__forceinline__ __device__ uchar4 make_normal_pixel( const float3& normal, float depth )  // pure
{
    return make_uchar4(
            static_cast<uint8_t>( clamp( normal.x, 0.0f, 1.0f ) *255.0f ),
            static_cast<uint8_t>( clamp( normal.y, 0.0f, 1.0f ) *255.0f ),
            static_cast<uint8_t>( clamp( normal.z, 0.0f, 1.0f ) *255.0f ),
            static_cast<uint8_t>( clamp( depth   , 0.0f, 1.0f ) *255.0f )
            );
}



/**
render : non-pure, uses params for viewpoint inputs and pixels output
-----------------------------------------------------------------------

**/

static __forceinline__ __device__ void render( const uint3& idx, const uint3& dim, quad2* prd )
{
    float2 d = 2.0f * make_float2(
            static_cast<float>(idx.x)/static_cast<float>(dim.x),
            static_cast<float>(idx.y)/static_cast<float>(dim.y)
            ) - 1.0f;

    //const bool yflip = true ;
    //if(yflip) d.y = -d.y ;

#ifdef DBG_PIDX
    bool dbg = idx.x == dim.x/2 && idx.y == dim.y/2 ;
    if(dbg) printf("//render.DBG_PIDX params.eye (%7.3f %7.3f %7.3f)\n", params.eye.x, params.eye.y, params.eye.z);
    if(dbg) printf("//render.DBG_PIDX params.U   (%7.3f %7.3f %7.3f)\n", params.U.x, params.U.y, params.U.z);
    if(dbg) printf("//render.DBG_PIDX params.V   (%7.3f %7.3f %7.3f)\n", params.V.x, params.V.y, params.V.z);
    if(dbg) printf("//render.DBG_PIDX params.W   (%7.3f %7.3f %7.3f)\n", params.W.x, params.W.y, params.W.z);
#endif

    const unsigned cameratype = params.cameratype ;
    const float3 dxyUV = d.x * params.U + d.y * params.V ;
    const float3 origin    = cameratype == 0u ? params.eye                     : params.eye + dxyUV    ;
    const float3 direction = cameratype == 0u ? normalize( dxyUV + params.W )  : normalize( params.W ) ;
    //                           cameratype 0u:perspective,                    1u:orthographic

    trace(
        params.handle,
        origin,
        direction,
        params.tmin,
        params.tmax,
        prd,
        params.vizmask
    );

    const float3* normal = prd->normal();
    float3 diddled_normal = normalize(*normal)*0.5f + 0.5f ;
    // "diddling" changes range of elements from -1.f:1.f to 0.f:1.f same as  (n+1.f)/2.f
    unsigned index = idx.y * params.width + idx.x ;



    float eye_z = -prd->distance()*dot(params.WNORM, direction) ;
    const float& A = params.ZPROJ.z ;
    const float& B = params.ZPROJ.w ;
    float zdepth = cameratype == 0u ? -(A + B/eye_z) : A*eye_z + B  ;  // cf SGLM::zdepth1

    if( prd->is_boundary_miss() ) zdepth = 0.999f ;

    uchar4 pixel = make_normal_pixel( diddled_normal, zdepth );

#ifdef DBG_PIDX
    if(dbg) printf("//render.DBG_PIDX pixel (%d %d %d %d) \n", pixel.x, pixel.y, pixel.z, pixel.w);
#endif

    params.pixels[index] = pixel ;
}


extern "C" __global__ void __raygen__rg()
{
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

#ifdef DBG_PIDX
    bool dbg = idx.x == dim.x/2 && idx.y == dim.y/2 ;
    if(dbg)  printf("//__raygen__rg.DBG_PIDX idx(%d,%d,%d) dim(%d,%d,%d)\n", idx.x, idx.y, idx.z, dim.x, dim.y, dim.z );
#endif

    quad2 prd ;
    prd.zero();

    render( idx, dim, &prd );
}

/**
__miss__ms
-------------

* missing "normal" is somewhat render specific and this is used for
  all raygenmode but Miss should never happen with real simulations
* Miss can happen with simple geometry testing however when shoot
  rays from outside the "world"

**/

extern "C" __global__ void __miss__ms()
{
    SOPTIX_MissData* ms = reinterpret_cast<SOPTIX_MissData*>( optixGetSbtDataPointer() );
    const unsigned ii_id = 0xffffffffu ;
    const unsigned gp_bd = 0xffffffffu ;
    const float lposcost = 0.f ;
    const float lposfphi = 0.f ;

    // printf("//__miss__ms ms.bg_color (%7.3f %7.3f %7.3f) \n", ms->bg_color.x, ms->bg_color.x, ms->bg_color.z );

    quad2* prd = SOPTIX_getPRD<quad2>();

    prd->q0.f.x = ms->bg_color.x ;  // HMM: thats setting the normal, so it will be diddled
    prd->q0.f.y = ms->bg_color.y ;
    prd->q0.f.z = ms->bg_color.z ;
    prd->q0.f.w = 0.f ;

    prd->q1.u.x = 0u ;
    prd->q1.u.y = 0u ;
    prd->q1.u.z = 0u ;
    prd->q1.u.w = 0u ;

    prd->set_globalPrimIdx_boundary_(gp_bd);
    prd->set_iindex_identity_(ii_id);

    prd->set_lpos(lposcost, lposfphi);  // __miss__ms.TRIANGLE


}

/**
__closesthit__ch
=================

optixGetInstanceIndex (aka iindex)
    0-based index within IAS

optixGetInstanceId (aka identity)
    user supplied instanceId,

optixGetPrimitiveIndex (aka prim_idx)
    CustomPrimitiveArray: local index of AABB within the GAS,
    TriangleArray: local index of triangle (HMM: within one buildInput?)

optixGetRayTmax
    In intersection and CH returns the current smallest reported hitT or the tmax passed into rtTrace
    if no hit has been reported


optixGetPrimitiveType
    returns OPTIX_PRIMITIVE_TYPE_TRIANGLE or OPTIX_PRIMITIVE_TYPE_CUSTOM


In general will need to branch between::

    OPTIX_BUILD_INPUT_TYPE_CUSTOM_PRIMITIVES
    OPTIX_BUILD_INPUT_TYPE_TRIANGLES

currently just handles triangles.

**/

extern "C" __global__ void __closesthit__ch()
{
    //OptixPrimitiveType type = optixGetPrimitiveType();
    //printf("//CH type %u \n", type );  hex(9521) = '0x2531'   OPTIX_PRIMITIVE_TYPE_TRIANGLE

    const SOPTIX_HitgroupData* hit_group_data = reinterpret_cast<SOPTIX_HitgroupData*>( optixGetSbtDataPointer() );
    const SOPTIX_TriMesh& mesh = hit_group_data->mesh ;

    //printf("//__closesthit__ch\n");

    const unsigned prim_idx = optixGetPrimitiveIndex();
    const float2   barys    = optixGetTriangleBarycentrics();

    uint3 tri = mesh.indice[ prim_idx ];
    const float3 P0 = mesh.vertex[ tri.x ];
    const float3 P1 = mesh.vertex[ tri.y ];
    const float3 P2 = mesh.vertex[ tri.z ];

    const float3 N0 = mesh.normal[ tri.x ];
    const float3 N1 = mesh.normal[ tri.y ];
    const float3 N2 = mesh.normal[ tri.z ];

    const float3 P = ( 1.0f-barys.x-barys.y)*P0 + barys.x*P1 + barys.y*P2;
    const float3 Ng = ( 1.0f-barys.x-barys.y)*N0 + barys.x*N1 + barys.y*N2; // guesss
    //const float3 Ng = cross( P1-P0, P2-P0 );

    const float3 N = normalize( optixTransformNormalFromObjectToWorldSpace( Ng ) );
    // HMM: could get normal by bary-weighting vertex normals ?

    unsigned iindex = optixGetInstanceIndex() ;
    unsigned identity = optixGetInstanceId() ;
    unsigned globalPrimIdx = 0u ;
    unsigned boundary = 0u ;
    // HMM: need to plant boundary in HitGroupData ?
    // cf CSGOptiX/Analytic: node->boundary();// all nodes of tree have same boundary

    float t = optixGetRayTmax() ;

    // cannot get Object frame ray_origin/direction in CH (only IS,AH)
    //const float3 ray_origin = optixGetObjectRayOrigin();
    //const float3 ray_direction = optixGetObjectRayDirection();
    //const float3 lpos = ray_origin + t*ray_direction  ;
    // HMM: could use P to give the local position ?

    float lposcost = normalize_cost(P); // scuda.h
    float lposfphi = normalize_fphi(P);

    quad2* prd = SOPTIX_getPRD<quad2>();

    prd->q0.f.x = N.x ;
    prd->q0.f.y = N.y ;
    prd->q0.f.z = N.z ;
    prd->q0.f.w = t ;

    prd->set_iindex_identity( iindex, identity ) ;
    prd->set_globalPrimIdx_boundary(globalPrimIdx, boundary) ;
    prd->set_lpos(lposcost, lposfphi);   // __closesthit__ch.TRIANGLE

}

/**
__intersection__is
====================

With inbuilt triangles there is no role for IS, the intersection
impl is provided by the Driver.

extern "C" __global__ void __intersection__is()
{
}

**/

